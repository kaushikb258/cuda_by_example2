#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

__global__ void addition(int *a, int *b, int *c)
{
   *c = *a + *b;
}


int main()
{
  int a, b, c;
  int *dev_a, *dev_b, *dev_c;
  int size = sizeof(int);

  hipError_t err;

  err = hipMalloc((void**)&dev_a, size);
  if(err != hipSuccess){
   cout<<"Error1 \n";
  }
  err = hipMalloc((void**)&dev_b, size);
  if(err != hipSuccess){
   cout<<"Error2 \n";
  }
  err = hipMalloc((void**)&dev_c, size);
  if(err != hipSuccess){
   cout<<"Error3 \n";
  }

  cout<<"enter value for a: \n";
  cin>>a;
  cout<<"enter value for b: \n";
  cin>>b;
  
  hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);  
  hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);  

  addition<<<1,1>>>(dev_a, dev_b, dev_c);
  hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

   hipFree(&dev_a);
   hipFree(&dev_b);
   hipFree(&dev_c);

   cout<<"sum of 2 numbers is: "<<c<<"\n";
   
   return 0;
}
