#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

#define N 20


__global__ void addition(int *a, int *b, int *c)
{
 int tid = blockIdx.x;
 if (tid < N) 
   c[tid] = a[tid] + b[tid];
}


int main()
{
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;
  int size = N*sizeof(int);
  int i;

  hipError_t err;

  err = hipMalloc((void**)&dev_a, size);
  if(err != hipSuccess){
   cout<<"Error1 \n";
  }
  err = hipMalloc((void**)&dev_b, size);
  if(err != hipSuccess){
   cout<<"Error2 \n";
  }
  err = hipMalloc((void**)&dev_c, size);
  if(err != hipSuccess){
   cout<<"Error3 \n";
  }

  for (i=0; i<N; i++){
   a[i] = -i;
   b[i] = i*i;
  }
  
  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);  
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);  

  addition<<<N,1>>>(dev_a, dev_b, dev_c);
  hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

  for (i=0; i<N; i++){
   cout<<a[i]<<" + "<<b[i]<<" = "<<c[i]<<"\n";
  }

   hipFree(dev_a);
   hipFree(dev_b);
   hipFree(dev_c);
   
   return 0;
}
