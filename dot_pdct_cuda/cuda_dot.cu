
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using namespace std;


#define imin(a,b) (a<b?a:b)

const int N = 33*1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);


// dot on the kernel
__global__ void dot(float *a, float *b, float *c)
{
 __shared__ float cache[threadsPerBlock];
 int cacheIndex = threadIdx.x;

 float temp = 0.0;
 for (int tid = threadIdx.x + blockIdx.x*blockDim.x; tid<N; tid += blockDim.x*gridDim.x) 
 {
  temp += a[tid]*b[tid]; 
 }
 
 cache[cacheIndex] = temp;

 __syncthreads();


 // reduction
 for (int i = blockDim.x/2; i>0; i /= 2)
 {
  if (cacheIndex < i) cache[cacheIndex] += cache[cacheIndex + i];
  __syncthreads();
 }

 if (threadIdx.x == 0) c[blockIdx.x] = cache[0];
}


// main fn
int main(void)
{
 float *a, *b, c, *partial_c;
 float *dev_a, *dev_b, *dev_partial_c;

 a = (float*)malloc(N*sizeof(float));
 b = (float*)malloc(N*sizeof(float)); 
 partial_c = (float*)malloc(blocksPerGrid*sizeof(float)); 

 hipMalloc((void**)&dev_a, N*sizeof(float));
 hipMalloc((void**)&dev_b, N*sizeof(float));
 hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float));

 for (int i=0; i<N; i++)
 {
  a[i] = i;
  b[i] = 2*i;
 }

 hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
 dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
 
 hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

 c = 0.0;
 for (int i=0; i<blocksPerGrid; i++)
 {
  c += partial_c[i];
 }  

 #define sum_squares(x) (x*(x+1)*(2*x+1)/6)
 cout<< "GPU value = "<<c<<" analytical value = "<<2*sum_squares((float)(N-1))<<endl;
 
 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(partial_c);
 
 free(a);
 free(b);
 free(partial_c);

}
