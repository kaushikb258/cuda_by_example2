#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <png.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "bitmap/bitmap_image.hpp"


using namespace std;

#define DIM 1000

struct cucomplex
{
 float r, i;
 __device__ cucomplex (float a, float b) : r(a), i(b) {}
 __device__ float mag2(void) 
 {
  return r*r + i*i;
 }
 __device__ cucomplex operator*(const cucomplex& a)
 {
  return cucomplex(r*a.r - i*a.i, i*a.r + r*a.i);
 }
 __device__ cucomplex operator+(const cucomplex& a)
 {
  return cucomplex(r+a.r,i+a.i);
 }
};


__device__ int julia(int x, int y)
{
 const float scale = 1.5;
 float jx = scale*(float)(DIM/2-x)/(DIM/2);
 float jy = scale*(float)(DIM/2-y)/(DIM/2);
 
 cucomplex c(-0.8, 0.156);
 cucomplex a(jx, jy);
 
 int i = 0;
 for (i=0; i<200; i++)
 {
  a = a*a + c;
  if (a.mag2() > 1000)
    return 0;
 }
 return 1;
}


__global__ void kernel(unsigned char *ptr)
{
 int x = blockIdx.x;
 int y = blockIdx.y;
 int offset = x + y*gridDim.x;

 int juliavalue = julia(x, y);
 ptr[offset*4 + 0] = 255*juliavalue;
 ptr[offset*4 + 1] = 0;
 ptr[offset*4 + 2] = 0;
 ptr[offset*4 + 3] = 255;
}


int main( void)
{
  bitmap_image image(DIM,DIM);
  cout<<"hello-4 \n";
  unsigned char* h_i;
  unsigned char *dev_bitmap;
  cout<<"hello-3 \n";
  hipMalloc((void**)&dev_bitmap, sizeof(int)*DIM*DIM); 
  cout<<"hello-2 \n";
  dim3 grid(DIM, DIM); 
  cout<<"hello-1 \n";
  kernel<<<grid,1>>>(dev_bitmap);
  cout<<"hello0 \n";
  h_i = (char*) malloc(DIM*DIM); 
  hipMemcpy(h_i, dev_bitmap, sizeof(int)*DIM*DIM, hipMemcpyDeviceToHost);
  cout<<"hello1 \n";
  image = *h_i;
  cout<<"hello2 \n"; 
  hipFree(dev_bitmap);
  image.save_image("test.bmp");
  return 0;  
}
