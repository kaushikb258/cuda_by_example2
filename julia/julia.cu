#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include "cpu_bitmap.h"


using namespace std;

#define DIM 1000


struct cucomplex
{
 float r, i;
 __device__ cucomplex (float a, float b) : r(a), i(b) {}
 __device__ float mag2(void) 
 {
  return r*r + i*i;
 }
 __device__ cucomplex operator*(const cucomplex& a)
 {
  return cucomplex(r*a.r - i*a.i, i*a.r + r*a.i);
 }
 __device__ cucomplex operator+(const cucomplex& a)
 {
  return cucomplex(r+a.r,i+a.i);
 }
};


__device__ int julia(int x, int y)
{
 const float scale = 1.5;
 float jx = scale*(float)(DIM/2-x)/(DIM/2);
 float jy = scale*(float)(DIM/2-y)/(DIM/2);
 
 cucomplex c(-0.8, 0.156);
 cucomplex a(jx, jy);
 
 int i = 0;
 for (i=0; i<200; i++)
 {
  a = a*a + c;
  if (a.mag2() > 1000)
    return 0;
 }
 return 1;
}


__global__ void kernel(unsigned char *ptr)
{
 int x = blockIdx.x;
 int y = blockIdx.y;
 int offset = x + y*gridDim.x;

 int juliavalue = julia(x, y);
 ptr[offset*4 + 0] = 255*juliavalue;
 ptr[offset*4 + 1] = 0;
 ptr[offset*4 + 2] = 0;
 ptr[offset*4 + 3] = 255;
}


int main()
{
  CPUBitmap bitmap(DIM, DIM);
  unsigned char *dev_bitmap;
  hipMalloc((void**)&dev_bitmap, bitmap.image_size()); 
  dim3 grid(DIM, DIM); 
  kernel<<<grid,1>>>(dev_bitmap);
  hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
  bitmap.display_and_exit();
  hipFree(dev_bitmap);
  return 0;
}
